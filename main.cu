#include "include/glad/glad.h"
#include <GL/glext.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#define GLFW_INCLUDE_NONE
#include <GLFW/glfw3.h>

#include <cuda_gl_interop.h>

#include "project_header.cuh"

void draw_world_in_terminal(const unsigned char *const world) {
  for (long i = 0; i < HEIGHT; ++i) {
    for (long j = 0; j < WIDTH; ++j) {
      unsigned char out;
      switch (world[j + i * WIDTH]) {
      case CELL_DEAD:
        out = ' ';
        break;
      case CELL_ALIVE:
        out = '+';
        break;
      }
      putchar(out);
    }
    putchar('\n');
  }
  puts("--------------------------------");
}

static bool g_space_just_pressed;
static void key_callback(GLFWwindow *window, int key, int scancode, int action,
                         int mods) {
  if (key == GLFW_KEY_SPACE && action == GLFW_PRESS) {
    g_space_just_pressed = true;
  }
}
static int g_current_window_width = 1024, g_current_window_height = 1024;
static void framebuffer_size_callback(GLFWwindow *window, int width,
                                      int height) {
  glViewport(0, 0, width, height);
  g_current_window_width = width;
  g_current_window_height = height;
}

// game of life, use shared memory so a 32x32 part will load into shared memory
// their values, and the middle 30x30 part will calculate but start by using
// global memory and divide thread blocks into chunks that calculate new grid
// array2D_set<<<GRIDDIM_WORLD, BLOCKDIM_WORLD>>>(d_world, WIDTH, CELL_DEAD);
// hipDeviceSynchronize();

#define SWAP_INTERVAL 0

int main() {
  srand(time(NULL));

  glfwInit();
  // glfwWindowHint(GLFW_DOUBLEBUFFER, GL_FALSE);
  glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
  glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 6);
  glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
  GLFWwindow *window = glfwCreateWindow(
      g_current_window_width, g_current_window_height, "CudaLife", NULL, NULL);
  glfwMakeContextCurrent(window);
  gladLoadGLLoader((GLADloadproc)glfwGetProcAddress);
  glViewport(0, 0, g_current_window_width, g_current_window_height);
  glfwSwapInterval(SWAP_INTERVAL);
  glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
  glfwSetKeyCallback(window, key_callback);

  unsigned int shader_program = get_program();

  unsigned int VAO;
  glGenVertexArrays(1, &VAO);

  unsigned int SSBO_first;
  glGenBuffers(1, &SSBO_first);
  glBindBuffer(GL_SHADER_STORAGE_BUFFER, SSBO_first);
  glBufferData(GL_SHADER_STORAGE_BUFFER, WORLD_BYTES, NULL, GL_DYNAMIC_DRAW);
  glBindBuffer(GL_SHADER_STORAGE_BUFFER, 0);
  glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 0, SSBO_first);
  randomize_world(SSBO_first);
  unsigned int SSBO_second;
  glGenBuffers(1, &SSBO_second);
  glBindBuffer(GL_SHADER_STORAGE_BUFFER, SSBO_second);
  glBufferData(GL_SHADER_STORAGE_BUFFER, WORLD_BYTES, NULL, GL_DYNAMIC_DRAW);
  glBindBuffer(GL_SHADER_STORAGE_BUFFER, 0);
  glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 1, SSBO_second);

  struct hipGraphicsResource *SSBO_CUDA_first;
  Chk(hipGraphicsGLRegisterBuffer(&SSBO_CUDA_first, SSBO_first,
                                   cudaGraphicsMapFlagsNone));
  struct hipGraphicsResource *SSBO_CUDA_second;
  Chk(hipGraphicsGLRegisterBuffer(&SSBO_CUDA_second, SSBO_second,
                                   cudaGraphicsMapFlagsNone));

  int u_pos_and_scale_location =
      glGetUniformLocation(shader_program, "u_pos_and_scale");
  float pos_x = 0.f, pos_y = 0.f, scale = 1.f;
  bool should_transform = true;

  double dt = 0.16, prev_time = 0.0;
  while (!glfwWindowShouldClose(window)) {
    g_space_just_pressed = false;
    glfwPollEvents();
    if (glfwGetKey(window, GLFW_KEY_R) == GLFW_PRESS) {
      randomize_world(SSBO_first);
    }
    float speed = 500.f * dt;
    if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS) {
      pos_x += speed;
    }
    if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS) {
      pos_x -= speed;
    }
    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS) {
      pos_y += speed;
    }
    if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS) {
      pos_y -= speed;
    }
    float scale_speed = 1 * dt * scale;
    if (glfwGetKey(window, GLFW_KEY_I) == GLFW_PRESS) {
      scale += scale_speed;
      pos_x += (pos_x + (float)(g_current_window_width >> 1)) * scale_speed *
               (1.f / scale);
      pos_y += (pos_y + (float)(g_current_window_height >> 1)) * scale_speed *
               (1.f / scale);
    }
    if (glfwGetKey(window, GLFW_KEY_O) == GLFW_PRESS) {
      scale -= scale_speed;
      pos_x -= (pos_x + (float)(g_current_window_width >> 1)) * scale_speed *
               (1.f / scale);
      pos_y -= (pos_y + (float)(g_current_window_height >> 1)) * scale_speed *
               (1.f / scale);
    }
    if (glfwGetKey(window, GLFW_KEY_T) == GLFW_PRESS) {
      terminate_all_life_in_world(SSBO_first);
    }
    if (g_space_just_pressed) {
      should_transform = !should_transform;
    }

    // mouse input
    {
      bool left_click =
          glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_LEFT) == GLFW_PRESS;
      bool right_click =
          glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_RIGHT) == GLFW_PRESS;
      if (left_click || right_click) {
        double x, y;
        glfwGetCursorPos(window, &x, &y);
        y = HEIGHT / 2 - y;
        // printf("pos: %f, %f, real: %f, %f\n", x, y, pos_x, pos_y);

        double real_x = (pos_x + x) / scale;
        double real_y = (pos_y + y) / scale;
        // printf("actual: %f, %f\n", real_x, real_y);
        {
          int cell_x = real_x, cell_y = real_y;
          world_set_cell(SSBO_first, cell_x, cell_y, left_click ? 1 : 0);
        }
      }
    }

    // Current bottleneck
    if (should_transform) { // transform world
      Chk(hipGraphicsMapResources(1, &SSBO_CUDA_first, 0));
      void *ssbo_first_mapped_to_cuda;
      Chk(hipGraphicsResourceGetMappedPointer(&ssbo_first_mapped_to_cuda, NULL,
                                               SSBO_CUDA_first));

      Chk(hipGraphicsMapResources(1, &SSBO_CUDA_second, 0));
      void *ssbo_second_mapped_to_cuda;
      Chk(hipGraphicsResourceGetMappedPointer(&ssbo_second_mapped_to_cuda,
                                               NULL, SSBO_CUDA_second));

      transform_world(
          (unsigned char *)ssbo_first_mapped_to_cuda, // IMPORTANT LINE
          (unsigned char *)ssbo_second_mapped_to_cuda);
      hipMemcpy(ssbo_first_mapped_to_cuda, ssbo_second_mapped_to_cuda,
                 WORLD_BYTES, hipMemcpyDeviceToDevice);

      hipGraphicsUnmapResources(1, &SSBO_CUDA_first);
      hipGraphicsUnmapResources(1, &SSBO_CUDA_second);
    }

    { // set delta time
      double time = glfwGetTime();
      dt = time - prev_time;
      prev_time = time;
    }
    glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);

    { // draw to screen
      glUseProgram(shader_program);
      glUniform4f(u_pos_and_scale_location, pos_x, pos_y, scale, scale);
      glBindVertexArray(VAO);
      glDrawArrays(GL_TRIANGLES, 0, 6);
    }

    { // set window title to framerate
      char buf[256];
      int written =
          snprintf(buf, sizeof(buf) - 1, "CudaLife: fps: %f", 1.f / dt);
      buf[written] = '\0';
      glfwSetWindowTitle(window, buf);
    }

    glfwSwapBuffers(window);
  }
  glfwDestroyWindow(window);
  glfwTerminate();
}
