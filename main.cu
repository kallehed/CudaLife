#include "hip/hip_runtime.h"
#include "raylib.h"
#include <cstdint>
#include <cstdio>
#include <cstdlib>
#include <ctime>

#define Chk(ans)                                                               \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

__global__ void array2D_set(char *a, const long width, const char val) {
  a[(threadIdx.x + blockIdx.x * blockDim.x) +
    (threadIdx.y + blockIdx.y * blockDim.y) * width] = val;
}

#define CELL_DEAD 0
#define CELL_ALIVE 1

__global__ void transform_cell(char *const world, const long width,
                               const long height) {
  const long x = blockIdx.x * blockDim.x + threadIdx.x;
  const long y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x == 0 || x == width - 1 || y == 0 || y == height - 1)
    return;
  const long place = x + y * width;
  const char cur_state = world[place];
  const char neighbors = world[place + 1] + world[place - 1] +
                         world[place + width] + world[place - width] +
                         world[place + 1 - width] + world[place + 1 + width] +
                         world[place - 1 + width] + world[place - 1 - width];
  char next_state;
  switch (neighbors) {
  case 0:
  case 1:
    next_state = CELL_DEAD;
    break;
  case 2:
    next_state = cur_state;
    break;
  case 3:
    next_state = CELL_ALIVE;
    break;
  default:
    next_state = CELL_DEAD;
  }
  __syncthreads();
  world[place] = next_state;
}

void draw_world_in_terminal(const char *const world, const long width,
                            const long height) {
  for (long i = 0; i < height; ++i) {
    for (long j = 0; j < width; ++j) {
      char out;
      switch (world[j + i * width]) {
      case CELL_DEAD:
        out = ' ';
        break;
      case CELL_ALIVE:
        out = '+';
        break;
      }
      putchar(out);
    }
    putchar('\n');
  }
  puts("--------------------------------");
}


static void draw_world_raylib(const char *const world, const long width,
                              const long height, const long window_width,
                              const long window_height, Texture2D render_tex, unsigned char *picture) {
  // Vector2 size = {window_width / (float)width, window_height /
  // (float)height};


  for (long i = 0; i < height; ++i) {
    for (long j = 0; j < width; ++j) {
      long idx = (j + i * width) * 4;
      // picture[idx + 3] = 255;
      char res = world[j + i * width];
      uint32_t bits = (res == 0) ? 0 : 0b11111111111111111111111111111111;

      *(uint32_t *)(&picture[idx]) = bits;
    }
  }
  UpdateTexture(render_tex, picture);
  DrawTexture(render_tex, 0, 0, WHITE);
}

static constexpr long WIDTH = 2048;
static constexpr long HEIGHT = 2048;
static constexpr long WORLD_BYTES = sizeof(char) * WIDTH * HEIGHT;
static constexpr dim3 BLOCKDIM_WORLD = dim3{32, 32, 1};
static constexpr dim3 GRIDDIM_WORLD = dim3{64, 64, 1};

static void transform_world(char *const d_world, const long width,
                            const long height) {
  transform_cell<<<GRIDDIM_WORLD, BLOCKDIM_WORLD>>>(d_world, width, height);
  hipDeviceSynchronize();
}

static void randomize_world(char *const h_world, char *const d_world,
                            const long width, const long height) {
  for (int i = 1; i < height - 1; ++i) {
    for (int j = 1; j < width - 1; ++j) {
      h_world[j + i * width] = rand() % 2;
    }
  }
  // upload to device from host
  hipMemcpy(d_world, h_world, WORLD_BYTES, hipMemcpyHostToDevice);
}

// game of life, use shared memory so a 32x32 part will load into shared memory
// their values, and the middle 30x30 part will calculate but start by using
// global memory and divide thread blocks into chunks that calculate new grid
// array2D_set<<<GRIDDIM_WORLD, BLOCKDIM_WORLD>>>(d_world, WIDTH, CELL_DEAD);
// hipDeviceSynchronize();
int main() {
  srand(time(NULL));
  char *h_world = (char *)malloc(WORLD_BYTES);
  char *d_world;
  hipMalloc(&d_world, WORLD_BYTES);

  randomize_world(h_world, d_world, WIDTH, HEIGHT);

  unsigned char *picture = (unsigned char *)malloc(WIDTH * HEIGHT * sizeof(char) * 4);

  // draw_world_in_terminal(h_world, WIDTH, HEIGHT);

  const long window_width = 2048, window_height = 2048;
  InitWindow(window_width, window_height, "cudalife");
  Texture2D render_tex;
  {
    Image img = GenImageColor(WIDTH, HEIGHT, WHITE);
    render_tex = LoadTextureFromImage(img);
    UnloadImage(img);
  }
  // SetTargetFPS(60);
  while (!WindowShouldClose()) // Detect window close button or ESC key
  {
    if (IsKeyPressed(KEY_R)) {
      randomize_world(h_world, d_world, WIDTH, HEIGHT);
    }
    // transform_world(d_world, WIDTH, HEIGHT);
    BeginDrawing();
    ClearBackground(BLACK);
    // copy from device to host
    hipMemcpy(h_world, d_world, WORLD_BYTES, hipMemcpyDeviceToHost);
    draw_world_raylib(h_world, WIDTH, HEIGHT, window_width, window_height, render_tex, picture);

    {
      float dt = GetFrameTime();
      char buf[256];
      int written = snprintf(buf, sizeof(buf) - 1, "fps: %f", 1.f / dt);
      buf[written] = '\0';
      DrawText(buf, 10, 10, 50, RED);
    }
    EndDrawing();
  }
  free(picture);
  UnloadTexture(render_tex);
  CloseWindow(); // Close window and OpenGL context
}
