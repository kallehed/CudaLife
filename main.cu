#include "hip/hip_runtime.h"
#include "include/glad/glad.h"
#include <GL/glext.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#define GLFW_INCLUDE_NONE
#include <GLFW/glfw3.h>

#include <cuda_gl_interop.h>

#define STR_INDIR(x) #x
#define STR(x) STR_INDIR(x)

#define Chk(ans)                                                               \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

__global__ void array2D_set(unsigned char *a, const long width,
                            const unsigned char val) {
  a[(threadIdx.x + blockIdx.x * blockDim.x) +
    (threadIdx.y + blockIdx.y * blockDim.y) * width] = val;
}

#define CELL_DEAD 0
#define CELL_ALIVE 1

// width and height of game of life cell 2D array
#define WIDTH 2048
#define HEIGHT 2048
static constexpr long WORLD_BYTES = sizeof(unsigned char) * WIDTH * HEIGHT;
static constexpr dim3 BLOCKDIM_WORLD =
    dim3{32, 32, 1}; // 32 * 32 is the maximum block
static constexpr dim3 GRIDDIM_WORLD = dim3{WIDTH / 32, HEIGHT / 32, 1};

__global__ void transform_cell(const unsigned char *const world,
                               unsigned char *write_world) {
  const long x = blockIdx.x * blockDim.x + threadIdx.x;
  const long y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x == 0 || x == WIDTH - 1 || y == 0 || y == HEIGHT - 1)
    return;
  const long place = x + y * WIDTH;
  const unsigned char cur_state = world[place];
  const unsigned char neighbors =
      world[place + 1] + world[place - 1] + world[place + WIDTH] +
      world[place - WIDTH] + world[place + 1 - WIDTH] +
      world[place + 1 + WIDTH] + world[place - 1 + WIDTH] +
      world[place - 1 - WIDTH];
  unsigned char next_state;
  switch (neighbors) {
  case 2:
    next_state = cur_state;
    break;
  case 3:
    next_state = CELL_ALIVE;
    break;
  default:
    next_state = CELL_DEAD;
  }
  write_world[place] = next_state;
}

void draw_world_in_terminal(const unsigned char *const world) {
  for (long i = 0; i < HEIGHT; ++i) {
    for (long j = 0; j < WIDTH; ++j) {
      unsigned char out;
      switch (world[j + i * WIDTH]) {
      case CELL_DEAD:
        out = ' ';
        break;
      case CELL_ALIVE:
        out = '+';
        break;
      }
      putchar(out);
    }
    putchar('\n');
  }
  puts("--------------------------------");
}

static void transform_world(const unsigned char *const read_world,
                            unsigned char *const write_world) {
  transform_cell<<<GRIDDIM_WORLD, BLOCKDIM_WORLD>>>(read_world, write_world);
  hipDeviceSynchronize();
}

// slow, copies using OpenGL, inits on CPU
static void randomize_world(unsigned int SSBO) {
  unsigned char *data = (unsigned char *)malloc(WORLD_BYTES);
  for (int i = 0; i < HEIGHT; ++i) {
    for (int j = 0; j < WIDTH; ++j) {
      unsigned char value;
      if (i == 0 || i == HEIGHT - 1 || j == 0 || j == WIDTH - 1) {
        value = 0;
      } else {
        value = rand() % 2;
      }
      data[i * WIDTH + j] = value;
    }
  }
  glBindBuffer(GL_SHADER_STORAGE_BUFFER, SSBO);
  glBufferSubData(GL_SHADER_STORAGE_BUFFER, 0, WORLD_BYTES, data);
  glBindBuffer(GL_SHADER_STORAGE_BUFFER, 0);
  free(data);
}
int g_current_window_width = 1024, g_current_window_height = 1024;
void framebuffer_size_callback(GLFWwindow *window, int width, int height) {
  glViewport(0, 0, width, height);
  g_current_window_width = width;
  g_current_window_height = height;
}

// game of life, use shared memory so a 32x32 part will load into shared memory
// their values, and the middle 30x30 part will calculate but start by using
// global memory and divide thread blocks into chunks that calculate new grid
// array2D_set<<<GRIDDIM_WORLD, BLOCKDIM_WORLD>>>(d_world, WIDTH, CELL_DEAD);
// hipDeviceSynchronize();

#define SWAP_INTERVAL 0

int main() {
  srand(time(NULL));

  glfwInit();
  // glfwWindowHint(GLFW_DOUBLEBUFFER, GL_FALSE);
  glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
  glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 6);
  glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
  GLFWwindow *window = glfwCreateWindow(
      g_current_window_width, g_current_window_height, "CudaLife", NULL, NULL);
  glfwMakeContextCurrent(window);
  gladLoadGLLoader((GLADloadproc)glfwGetProcAddress);
  glViewport(0, 0, g_current_window_width, g_current_window_height);
  glfwSwapInterval(SWAP_INTERVAL);
  glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);

  const char *const vertex_shader_source =
      "#version 460\n"
      "float x_pos[6] = float[6](-1.f, -1.f, 1.f, -1.f, 1.f, 1.f); \n"
      "float y_pos[6] = float[6](-1.f, 1.f, 1.f, -1.f, 1.f, -1.f); \n"
      "void main()\n"
      "{\n"
      " float x = x_pos[gl_VertexID];"
      " float y = y_pos[gl_VertexID];"
      " gl_Position = vec4(x, y, 0.f, 1.0);\n"
      "}\0";
  unsigned int vertex_shader = glCreateShader(GL_VERTEX_SHADER);
  glShaderSource(vertex_shader, 1, &vertex_shader_source, NULL);
  glCompileShader(vertex_shader);
  {
    int success;
    char infoLog[512];
    glGetShaderiv(vertex_shader, GL_COMPILE_STATUS, &success);
    if (!success) {
      glGetShaderInfoLog(vertex_shader, 512, NULL, infoLog);
      printf("ERROR::SHADER::VERTEX::COMPILATION_FAILED\n %s\n", infoLog);
    }
  }
  // clang-format off
  const char *const fragment_shader_source =
      "#version 460 core\n"
      "out vec4 FragColor;\n"
      "uniform vec4 u_pos_and_scale;"
      "layout (std430, binding = 0) buffer Colors {\n"
      "  uint color[];\n" 
      "};\n"
      "void main() {\n"
      "  uint x = uint((gl_FragCoord.x + u_pos_and_scale.x)/u_pos_and_scale.z);\n"
      "  uint y = uint((gl_FragCoord.y + u_pos_and_scale.y)/u_pos_and_scale.w);\n"
      "  uint idx = x + y * " STR(WIDTH) ";\n"
      "  uint block = idx / 4;\n"
      "  uint byte = idx % 4;\n"
      "  uint col4 = color[block];\n"
      "  uint mask = (0x000000FF << (byte * 8));\n"
      "  uint colbool =  mask & col4;\n"
      "  float col = float(colbool);\n"
      "  if (x >= " STR(WIDTH) " || y >= " STR(HEIGHT) ") {col = 0.5;}"
      "  FragColor = vec4(vec3(float(col)), 1.0f);\n"
      "}\n";
  // clang-format on
  unsigned int fragment_shader;
  fragment_shader = glCreateShader(GL_FRAGMENT_SHADER);
  glShaderSource(fragment_shader, 1, &fragment_shader_source, NULL);
  glCompileShader(fragment_shader);
  {
    int success;
    char infoLog[512];
    glGetShaderiv(fragment_shader, GL_COMPILE_STATUS, &success);
    if (!success) {
      glGetShaderInfoLog(fragment_shader, 512, NULL, infoLog);
      printf("ERROR::SHADER::FRAGMENT::COMPILATION_FAILED\n %s\n", infoLog);
    }
  }
  unsigned int shader_program;
  shader_program = glCreateProgram();
  glAttachShader(shader_program, vertex_shader);
  glAttachShader(shader_program, fragment_shader);
  glLinkProgram(shader_program);
  {
    int success;
    char infoLog[512];
    glGetProgramiv(shader_program, GL_LINK_STATUS, &success);
    if (!success) {
      glGetProgramInfoLog(shader_program, 512, NULL, infoLog);
      printf("ERROR::PROGRAM::COMPILATION_FAILED\n %s\n", infoLog);
    }
  }
  glDeleteShader(vertex_shader);
  glDeleteShader(fragment_shader);

  unsigned int VAO;
  glGenVertexArrays(1, &VAO);

  unsigned int SSBO_first;
  glGenBuffers(1, &SSBO_first);
  glBindBuffer(GL_SHADER_STORAGE_BUFFER, SSBO_first);
  glBufferData(GL_SHADER_STORAGE_BUFFER, WORLD_BYTES, NULL, GL_DYNAMIC_DRAW);
  glBindBuffer(GL_SHADER_STORAGE_BUFFER, 0);
  glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 0, SSBO_first);
  randomize_world(SSBO_first);
  unsigned int SSBO_second;
  glGenBuffers(1, &SSBO_second);
  glBindBuffer(GL_SHADER_STORAGE_BUFFER, SSBO_second);
  glBufferData(GL_SHADER_STORAGE_BUFFER, WORLD_BYTES, NULL, GL_DYNAMIC_DRAW);
  glBindBuffer(GL_SHADER_STORAGE_BUFFER, 0);
  glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 1, SSBO_second);

  struct hipGraphicsResource *SSBO_CUDA_first;
  Chk(hipGraphicsGLRegisterBuffer(&SSBO_CUDA_first, SSBO_first, cudaGraphicsMapFlagsNone));
  struct hipGraphicsResource *SSBO_CUDA_second;
  Chk(hipGraphicsGLRegisterBuffer(&SSBO_CUDA_second, SSBO_second, cudaGraphicsMapFlagsNone));

  int u_pos_and_scale_location =
      glGetUniformLocation(shader_program, "u_pos_and_scale");
  float pos_x = 0.f, pos_y = 0.f, scale = 1.f;

  double dt = 0.16, prev_time = 0.0;
  while (!glfwWindowShouldClose(window)) {
    glfwPollEvents();
    if (glfwGetKey(window, GLFW_KEY_R) == GLFW_PRESS) {
      randomize_world(SSBO_first);
    }
    float speed = 500.f * dt;
    if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS) {
      pos_x += speed;
    }
    if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS) {
      pos_x -= speed;
    }
    if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS) {
      pos_y += speed;
    }
    if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS) {
      pos_y -= speed;
    }
    float scale_speed = 1 * dt * scale;
    if (glfwGetKey(window, GLFW_KEY_I) == GLFW_PRESS) {
      scale += scale_speed;
      pos_x += (pos_x + (float)(g_current_window_width >> 1)) * scale_speed *
               (1.f / scale);
      pos_y += (pos_y + (float)(g_current_window_height >> 1)) * scale_speed *
               (1.f / scale);
    }
    if (glfwGetKey(window, GLFW_KEY_O) == GLFW_PRESS) {
      scale -= scale_speed;
      pos_x -= (pos_x + (float)(g_current_window_width >> 1)) * scale_speed *
               (1.f / scale);
      pos_y -= (pos_y + (float)(g_current_window_height >> 1)) * scale_speed *
               (1.f / scale);
    }

    // Current bottleneck
    // if (glfwGetKey(window, GLFW_KEY_SPACE) == GLFW_PRESS) {
    if (!false) {
      Chk(hipGraphicsMapResources(1, &SSBO_CUDA_first, 0));
      void *ssbo_first_mapped_to_cuda;
      Chk(hipGraphicsResourceGetMappedPointer(&ssbo_first_mapped_to_cuda, NULL,
                                               SSBO_CUDA_first));

      Chk(hipGraphicsMapResources(1, &SSBO_CUDA_second, 0));
      void *ssbo_second_mapped_to_cuda;
      Chk(hipGraphicsResourceGetMappedPointer(&ssbo_second_mapped_to_cuda, NULL,
                                               SSBO_CUDA_second));

      transform_world((unsigned char *)ssbo_first_mapped_to_cuda, (unsigned char *)ssbo_second_mapped_to_cuda);
      hipMemcpy(ssbo_first_mapped_to_cuda, ssbo_second_mapped_to_cuda, WORLD_BYTES, hipMemcpyDeviceToDevice);
      hipGraphicsUnmapResources(1, &SSBO_CUDA_first);
      hipGraphicsUnmapResources(1, &SSBO_CUDA_second);
    }

    {
      double time = glfwGetTime();
      dt = time - prev_time;
      prev_time = time;
    }
    glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
    glClear(GL_COLOR_BUFFER_BIT);

    if (!false) {
      glUseProgram(shader_program);
      glUniform4f(u_pos_and_scale_location, pos_x, pos_y, scale, scale);
      glBindVertexArray(VAO);
      glDrawArrays(GL_TRIANGLES, 0, 6);
    }

    if (!false) {
      char buf[256];
      int written =
          snprintf(buf, sizeof(buf) - 1, "CudaLife: fps: %f", 1.f / dt);
      buf[written] = '\0';
      glfwSetWindowTitle(window, buf);
    }

    glfwSwapBuffers(window);
  }
  glfwDestroyWindow(window);
  glfwTerminate();
}
