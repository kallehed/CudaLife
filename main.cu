
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>

#define Chk(ans)                                                               \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

__global__ void array2D_set(char *a, const long width, const char val) {
  a[(threadIdx.x + blockIdx.x * blockDim.x) +
    (threadIdx.y + blockIdx.y * blockDim.y) * width] = val;
}

#define CELL_DEAD 0
#define CELL_ALIVE 1

__global__ void transform_cell(char *const world, const long width,
                               const long height) {
  const long x = blockIdx.x * blockDim.x + threadIdx.x;
  const long y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x == 0 || x == width - 1 || y == 0 || y == height - 1)
    return;
  const long place = x + y * width;
  const char cur_state = world[place];
  const char neighbors = world[place + 1] + world[place - 1] +
                         world[place + width] + world[place - width] +
                         world[place + 1 - width] + world[place + 1 + width] +
                         world[place - 1 + width] + world[place - 1 - width];
  char next_state;
  switch (neighbors) {
  case 0:
  case 1:
    next_state = CELL_DEAD;
    break;
  case 2:
    next_state = cur_state;
    break;
  case 3:
    next_state = CELL_ALIVE;
    break;
  default:
    next_state = CELL_DEAD;
  }
  world[place] = next_state;
}

void draw_world_in_terminal(const char *const world, const long width,
                            const long height) {
  for (long i = 0; i < height; ++i) {
    for (long j = 0; j < width; ++j) {
      char out;
      switch (world[j + i * width]) {
        case CELL_DEAD:
          out = ' ';
          break;
        case CELL_ALIVE:
          out = '+';
          break;
      }
      putchar(out);
    }
    putchar('\n');
  }
}

static constexpr long WIDTH = 32;
static constexpr long HEIGHT = 32;
static constexpr long WORLD_BYTES = sizeof(char) * WIDTH * HEIGHT;

static constexpr dim3 BLOCKDIM_WORLD = dim3{32, 32, 1};
static constexpr dim3 GRIDDIM_WORLD = dim3{1, 1, 1};

// game of life, use shared memory so a 32x32 part will load into shared memory
// their values, and the middle 30x30 part will calculate but start by using
// global memory and divide thread blocks into chunks that calculate new grid

int main() {
  srand(time(NULL));
  char *h_world = (char *)malloc(WORLD_BYTES);
  char *d_world;
  hipMalloc(&d_world, WORLD_BYTES);
  // array2D_set<<<GRIDDIM_WORLD, BLOCKDIM_WORLD>>>(d_world, WIDTH, CELL_DEAD);
  // cudaDeviceSynchronize();

  // set host to random
  for (int i = 1; i < HEIGHT - 1; ++i) {
    for (int j = 1; j < WIDTH - 1; ++j) {
      h_world[j + i * WIDTH] = rand() % 2;
    }
  }
  // upload to device from host
  hipMemcpy(d_world, h_world, WORLD_BYTES, hipMemcpyHostToDevice);

  for (int iter = 0; iter < 10000; ++iter) {
    transform_cell<<<GRIDDIM_WORLD, BLOCKDIM_WORLD>>>(d_world, WIDTH, HEIGHT);
    hipDeviceSynchronize();

    // copy back to host
    hipMemcpy(h_world, d_world, WORLD_BYTES, hipMemcpyDeviceToHost);
    draw_world_in_terminal(h_world, WIDTH, HEIGHT);
    puts("--------------------------------");
  }

  float total_error = 0;
  for (long i = 0; i < HEIGHT; ++i) {
    for (long j = 0; j < WIDTH; ++j) {
      total_error += abs(h_world[j + i * WIDTH]);
    }
    // std::cout << " " << h_a[i] << " ";
  }
  printf("total life: %f\n" , (float)total_error);
}
